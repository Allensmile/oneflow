#include "hip/hip_runtime.h"
#include "oneflow/core/kernel/cuda_ring_all_reduce_kernel.h"
#include "oneflow/core/kernel/kernel_util.cuh"
#include <>

namespace oneflow {

constexpr int32_t PACK_SIZE = sizeof(ulong2);
constexpr int32_t PACK_ALIGN = alignof(ulong2);
constexpr int32_t NUM_WARP = 8;
constexpr int32_t NUM_THREAD_PER_WARP = 32;
constexpr int32_t NUM_THREAD = NUM_THREAD_PER_WARP * NUM_WARP;
constexpr int32_t NUM_LINE_PER_CHUNK = 32;
constexpr int32_t NUM_PACK_PER_LINE_PER_THREAD = 8;
constexpr int32_t NUM_PACK_PER_LINE_PER_WARP = NUM_PACK_PER_LINE_PER_THREAD * NUM_THREAD_PER_WARP;
constexpr int32_t NUM_PACK_PER_LINE = NUM_PACK_PER_LINE_PER_WARP * NUM_WARP;
constexpr int32_t LINE_SIZE = NUM_PACK_PER_LINE * PACK_SIZE;
constexpr int32_t CHUNK_SIZE = LINE_SIZE * NUM_LINE_PER_CHUNK;

namespace {

template<typename T>
__global__ void AllReduceGpu(CudaRingAllReduceArg<T> arg) {}

template<typename T>
__global__ void SendGpu(CudaRingAllReduceArg<T> arg) {
  const int32_t thread_id = threadIdx.x;
  const int32_t block_id = blockIdx.x;
  T* send = arg.send[block_id];
  const T* src = arg.src[block_id];
  const int64_t num_elem = arg.num_elem[block_id];
  static_assert(PACK_SIZE % sizeof(T) == 0, "");
  const int32_t num_elem_per_line = LINE_SIZE / sizeof(T);
  const int64_t num_line = num_elem / num_elem_per_line;
  const int32_t warp_id = thread_id / NUM_THREAD_PER_WARP;
  const int32_t lane_id = thread_id % NUM_THREAD_PER_WARP;
  ulong2 line[NUM_PACK_PER_LINE_PER_THREAD];
  const int32_t offset = warp_id * NUM_PACK_PER_LINE_PER_WARP + lane_id;
  ulong2* dst_pack_ptr = reinterpret_cast<ulong2*>(send) + offset;
  const ulong2* src_pack_ptr = reinterpret_cast<const ulong2*>(src) + offset;
  for (int64_t l = 0; l < num_line; ++l) {
#pragma unroll
    for (int32_t p = 0; p < NUM_PACK_PER_LINE_PER_THREAD; ++p) {
      line[p] = *(src_pack_ptr + p * NUM_THREAD_PER_WARP);
    }
#pragma unroll
    for (int32_t p = 0; p < NUM_PACK_PER_LINE_PER_THREAD; ++p) {
      *(dst_pack_ptr + p * NUM_THREAD_PER_WARP) = line[p];
    }
    dst_pack_ptr += NUM_PACK_PER_LINE;
    src_pack_ptr += NUM_PACK_PER_LINE;
  }
}

}  // namespace

template<typename T>
void CudaRingAllReduceKernelUtil<T>::AllReduce(DeviceCtx* ctx, CudaRingAllReduceArg<T> arg) {
  AllReduceGpu<<<arg.num_rings, 256, 0, ctx->hip_stream()>>>(arg);
}

template<typename T>
void CudaRingAllReduceKernelUtil<T>::Send(DeviceCtx* ctx, CudaRingAllReduceArg<T> arg) {
  SendGpu<<<arg.num_rings, 256, 0, ctx->hip_stream()>>>(arg);
}

#define INSTANTIATE_CUDA_RING_ALL_REDUCE_KERNEL_UTIL(type_cpp, type_proto) \
  template struct CudaRingAllReduceKernelUtil<type_cpp>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_CUDA_RING_ALL_REDUCE_KERNEL_UTIL, FLOATING_DATA_TYPE_SEQ)

}  // namespace oneflow
