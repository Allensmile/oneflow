#include "hip/hip_runtime.h"
#include "oneflow/core/kernel/batch_gather_kernel_util.h"
#include "oneflow/core/kernel/kernel_util.cuh"
#include <assert.h>

namespace oneflow {

namespace {

Shape GetFlatShape(const ShapeView& shape, const int64_t axis) {
  CHECK_GT(shape.NumAxes(), 0);
  CHECK_GE(axis, 0);
  CHECK_LT(axis, shape.NumAxes());
  return Shape({shape.Count(0, axis), shape.At(axis), shape.Count(axis + 1)});
}

template<typename K, typename IDX>
__device__ int64_t GetInOffset(const IDX out_offset, const K* indices, const IDX indices_num,
                               const IDX instance_size, const IDX gather_dim_size,
                               const IDX in_batch_size, const IDX out_batch_size) {
  const IDX batch_idx = out_offset / out_batch_size;
  const IDX indices_idx = out_offset % out_batch_size / instance_size;
  const IDX inner_idx = out_offset % instance_size;
  const K idx = indices[batch_idx * indices_num + indices_idx];
  assert(idx >= 0 && idx < gather_dim_size);
  return batch_idx * in_batch_size + idx * instance_size + inner_idx;
}

template<typename T, typename K, typename IDX>
__global__ void BatchGatherForwardGpu(const IDX elem_cnt, const T* in, const K* indices,
                                      const IDX indices_num, const IDX instance_size,
                                      const IDX gather_dim_size, T* out) {
  const IDX in_batch_size = gather_dim_size * instance_size;
  const IDX out_batch_size = indices_num * instance_size;
  CUDA_1D_KERNEL_LOOP_T(IDX, i, elem_cnt) {
    out[i] = in[GetInOffset<K, IDX>(i, indices, indices_num, instance_size, gather_dim_size,
                                    in_batch_size, out_batch_size)];
  }
}

}  // namespace

template<typename T, typename K>
struct BatchGatherKernelUtilImpl<DeviceType::kGPU, T, K> final {
  static void Forward(DeviceCtx* ctx, const T* in, const K* indices, const Shape& flat_out_shape,
                      const int64_t gather_dim_size, T* out);
};

template<typename T, typename K>
void BatchGatherKernelUtilImpl<DeviceType::kGPU, T, K>::Forward(DeviceCtx* ctx, const T* in,
                                                                const K* indices,
                                                                const Shape& flat_out_shape,
                                                                const int64_t gather_dim_size,
                                                                T* out) {
  const int64_t batch_num = flat_out_shape.At(0);
  const int64_t indices_num = flat_out_shape.At(1);
  const int64_t instance_size = flat_out_shape.At(2);
  const int64_t out_elem_cnt = batch_num * indices_num * instance_size;
  const int64_t in_elem_cnt = batch_num * gather_dim_size * instance_size;
  if (std::max(out_elem_cnt, in_elem_cnt) > GetMaxVal<int32_t>() / 2) {
    BatchGatherForwardGpu<T, K, int64_t>
        <<<BlocksNum4ThreadsNum(out_elem_cnt), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
            out_elem_cnt, in, indices, indices_num, instance_size, gather_dim_size, out);
  } else {
    BatchGatherForwardGpu<T, K, int32_t>
        <<<BlocksNum4ThreadsNum(out_elem_cnt), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
            out_elem_cnt, in, indices, indices_num, instance_size, gather_dim_size, out);
  }
}

#define INSTANTIATE_BATCH_GATHER_KERNEL_UTIL_IMPL_GPU(in_type_pair, index_type_pair)          \
  template struct BatchGatherKernelUtilImpl<DeviceType::kGPU, OF_PP_PAIR_FIRST(in_type_pair), \
                                            OF_PP_PAIR_FIRST(index_type_pair)>;
OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_BATCH_GATHER_KERNEL_UTIL_IMPL_GPU,
                                 FLOATING_DATA_TYPE_SEQ, INDEX_DATA_TYPE_SEQ);
#undef INSTANTIATE_BATCH_GATHER_KERNEL_UTIL_IMPL_GPU

}  // namespace oneflow
