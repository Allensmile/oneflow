#include "hip/hip_runtime.h"
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/kernel/lazy_adam_model_update_kernel.h"
#include "oneflow/core/kernel/normal_model_update_kernel.cuh"

namespace oneflow {

namespace {

template<typename T>
__global__ void UpdateModelGpu(int64_t n, const float* learning_rate, T l1, T l2, T beta1, T beta2,
                               T epsilon, const T* beta1_t, const T* beta2_t, T* model_diff,
                               T* model, T* m, T* v, const int64_t* train_step) {
  const T beta1_t_v = pow(beta1, *train_step + 1);
  const T beta2_t_v = pow(beta2, *train_step + 1);
  const float local_learning_rate = *learning_rate * sqrt(1 - (beta2_t_v)) / (1 - (beta1_t_v));
  CUDA_1D_KERNEL_LOOP_T(int64_t, i, n) {
    if (abs(model_diff[i]) < 1e-12) { continue; }
    T reg_diff = RegDiff(model_diff[i], l1, l2, model[i]);
    m[i] = beta1 * m[i] + (1 - beta1) * reg_diff;
    v[i] = beta2 * v[i] + (1 - beta2) * reg_diff * reg_diff;
    model[i] = model[i] - local_learning_rate * m[i] / (sqrt(v[i]) + epsilon);
  }
}

}  // namespace

template<typename T>
class LazyAdamMdUpdateKernelUtil<DeviceType::kGPU, T> final {
 public:
  static void UpdateModel(DeviceCtx* ctx, int64_t n, const float* learning_rate, T l1, T l2,
                          T beta1, T beta2, T epsilon, const int64_t* train_step, T* beta1_t,
                          T* beta2_t, T* model_diff, T* model, T* m, T* v) {
    const int32_t num_threads = static_cast<int32_t>(std::max<int64_t>(n, 512));
    const int32_t num_blocks = static_cast<int32_t>(std::max<int64_t>((n - 1) / 512 + 1, 512));
    UpdateModelGpu<T><<<num_blocks, num_threads, 0, ctx->hip_stream()>>>(
        n, learning_rate, l1, l2, beta1, beta2, epsilon, beta1_t, beta2_t, model_diff, model, m, v,
        train_step);
  }
};

#define INSTANTIATE_GPU_KERNEL_UTIL(type_cpp, type_proto) \
  template class LazyAdamMdUpdateKernelUtil<DeviceType::kGPU, type_cpp>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_GPU_KERNEL_UTIL, FLOATING_DATA_TYPE_SEQ)

}  // namespace oneflow
