#include "hip/hip_runtime.h"
#include "oneflow/core/kernel/cuda_copy_peer_kernel_util.h"
#include "oneflow/core/kernel/kernel_util.cuh"

constexpr int32_t PACK_SIZE = sizeof(ulong2);
constexpr int32_t PACK_ALIGN = alignof(ulong2);
constexpr int32_t NUM_THREAD = 1024;
constexpr int32_t NUM_STEP_PER_CHUNK = 16;
constexpr int32_t STEP_SIZE = NUM_THREAD * PACK_SIZE;
constexpr int32_t CHUNK_SIZE = STEP_SIZE * NUM_STEP_PER_CHUNK;
constexpr int32_t DEFAULT_CHUNK_BUF_CAP = 2;
constexpr int32_t WARP_SIZE = 32;
constexpr int32_t MAX_NUM_BLOCK = 2;

namespace oneflow {

namespace {

__forceinline__ __device__ int32_t DivUp(int32_t n, int32_t val) { return (n + val - 1) / val; }

__forceinline__ __device__ void Fetch(ulong2& v, const ulong2* p) {
  // clang-format off
  asm volatile("ld.volatile.global.v2.u64 {%0,%1}, [%2];" : "=l"(v.x), "=l"(v.y) : "l"(p) : "memory");
  // clang-format on
}

__forceinline__ __device__ void Store(ulong2* p, ulong2& v) {
  // clang-format off
  asm volatile("st.volatile.global.v2.u64 [%0], {%1,%2};" :: "l"(p), "l"(v.x), "l"(v.y) : "memory");
  // clang-format on
}

__forceinline__ __device__ void FetchStore(ulong2* dst, const ulong2* src) {
  ulong2 v;
  Fetch(v, src);
  Store(dst, v);
}

__forceinline__ __device__ void FetchStore(void* dst, const void* src) {
  FetchStore(reinterpret_cast<ulong2*>(dst), reinterpret_cast<const ulong2*>(src));
}

__forceinline__ __device__ void CopyChunk(void* dst, const void* src, const int32_t thread_id) {
  dst = (unsigned char*)(dst) + thread_id * PACK_SIZE;
  src = (unsigned char*)(src) + thread_id * PACK_SIZE;
#pragma unroll
  for (int32_t i = 0; i < NUM_STEP_PER_CHUNK; ++i) {
    FetchStore(dst, src);
    dst = (unsigned char*)(dst) + STEP_SIZE;
    src = (const unsigned char*)(src) + STEP_SIZE;
  }
}

__forceinline__ __device__ void CopyPartialChunk(void* dst, const void* src, const int32_t size,
                                                 const int32_t thread_id) {
  int32_t offset = thread_id * PACK_SIZE;
  for (int32_t i = 0; i < NUM_STEP_PER_CHUNK; ++i) {
    if (offset < size) {
      FetchStore((unsigned char*)(dst) + offset, (const unsigned char*)(src) + offset);
    }
    offset += STEP_SIZE;
  }
}

__forceinline__ __device__ void Send(const void* src, const int32_t size, const int32_t thread_id,
                                     void* buf_ptr, const int32_t buf_cap,
                                     volatile int32_t* send_cnt_ptr,
                                     volatile int32_t* recv_cnt_ptr) {
  const int32_t num_chunk = DivUp(size, CHUNK_SIZE);
  int32_t remaining = size;
  if (thread_id == 0) {
    while (*recv_cnt_ptr != 0) {}
    while (*send_cnt_ptr != 0) {}
  }
  __syncthreads();
  for (int32_t chunk = 0; chunk < num_chunk; ++chunk) {
    if (thread_id == WARP_SIZE) {
      while (chunk - *recv_cnt_ptr >= buf_cap) {}
    }
    __syncthreads();
    void* cur_buf_ptr = (unsigned char*)buf_ptr + (chunk % buf_cap) * CHUNK_SIZE;
    if (remaining >= CHUNK_SIZE) {
      CopyChunk(cur_buf_ptr, src, thread_id);
    } else {
      CopyPartialChunk(cur_buf_ptr, src, remaining, thread_id);
    }
    remaining -= CHUNK_SIZE;
    src = (const unsigned char*)(src) + CHUNK_SIZE;
    __threadfence_system();
    __syncthreads();
    if (thread_id == 0) { *send_cnt_ptr = chunk + 1; }
  }
}

__forceinline__ __device__ void Recv(void* dst, const int32_t size, const int32_t thread_id,
                                     const void* buf_ptr, const int32_t buf_cap,
                                     volatile int32_t* send_cnt_ptr,
                                     volatile int32_t* recv_cnt_ptr) {
  const int32_t num_chunk = DivUp(size, CHUNK_SIZE);
  int32_t remaining = size;
  for (int32_t chunk = 0; chunk < num_chunk; ++chunk) {
    if (thread_id == WARP_SIZE) {
      while (*send_cnt_ptr <= chunk) {}
    }
    __syncthreads();
    void* cur_buf_ptr = (unsigned char*)buf_ptr + (chunk % buf_cap) * CHUNK_SIZE;
    if (remaining >= CHUNK_SIZE) {
      CopyChunk(dst, cur_buf_ptr, thread_id);
    } else {
      CopyPartialChunk(dst, cur_buf_ptr, remaining, thread_id);
    }
    remaining -= CHUNK_SIZE;
    dst = (unsigned char*)(dst) + CHUNK_SIZE;
    __syncthreads();
    if (thread_id == 0) { *recv_cnt_ptr = chunk + 1; }
  }
  if (thread_id == 0) {
    *recv_cnt_ptr = 0;
    *send_cnt_ptr = 0;
  }
}

__launch_bounds__(NUM_THREAD) __global__
    void Copy(void* dst, const void* src, const int32_t size, void* buf_ptr, const int32_t buf_cap,
              int32_t* send_cnt_ptr, int32_t* recv_cnt_ptr, bool send_or_recv) {
  const int32_t block_id = blockIdx.x;
  const int32_t num_block = gridDim.x;
  const int32_t thread_id = threadIdx.x;
  const int32_t block_size = DivUp(size / PACK_SIZE, num_block) * PACK_SIZE;
  void* this_block_dst = reinterpret_cast<unsigned char*>(dst) + block_size * block_id;
  const void* this_block_src = reinterpret_cast<const unsigned char*>(src) + block_size * block_id;
  const int32_t this_block_size =
      (block_id + 1) * block_size <= size ? block_size : max(0, size - block_id * block_size);
  void* this_buf_ptr = reinterpret_cast<unsigned char*>(buf_ptr) + CHUNK_SIZE * buf_cap * block_id;
  int32_t* this_send_cnt_ptr = send_cnt_ptr + block_id;
  int32_t* this_recv_cnt_ptr = recv_cnt_ptr + block_id;
  if (send_or_recv) {
    Send(this_block_src, this_block_size, thread_id, this_buf_ptr, buf_cap, this_send_cnt_ptr,
         this_recv_cnt_ptr);
  } else {
    Recv(this_block_dst, this_block_size, thread_id, this_buf_ptr, buf_cap, this_send_cnt_ptr,
         this_recv_cnt_ptr);
  }
}

__global__ void Null() {}

}  // namespace

struct CudaCopyPeerCtx {
  int32_t dst_dev_id;
  int32_t src_dev_id;
  hipStream_t recv_stream;
  hipStream_t send_stream;
  int32_t num_block;
  int32_t* recv_cnt_ptr;
  int32_t* send_cnt_ptr;
  void* buf_ptr;
  int32_t buf_cap;
  bool p2p_enabled;
};

void CudaCopyPeerKernelUtil::CtxCreate(CudaCopyPeerCtx** ctx, int32_t dst_dev_id,
                                       int32_t src_dev_id, hipStream_t recv_stream) {
  *ctx = new CudaCopyPeerCtx();
  (*ctx)->dst_dev_id = dst_dev_id;
  (*ctx)->src_dev_id = src_dev_id;
  (*ctx)->recv_stream = recv_stream;

  WithCudaDevice(dst_dev_id, [ctx]() {
    int32_t can_access;
    CudaCheck(hipDeviceCanAccessPeer(&can_access, (*ctx)->dst_dev_id, (*ctx)->src_dev_id));
    if (can_access) {
      hipError_t error = hipDeviceEnablePeerAccess((*ctx)->src_dev_id, 0);
      if (error != hipErrorPeerAccessAlreadyEnabled) { CudaCheck(error); }
      (*ctx)->p2p_enabled = true;
    } else {
      (*ctx)->p2p_enabled = false;
    }
  });
  if (!(*ctx)->p2p_enabled) {
    WithCudaDevice(src_dev_id, [ctx]() { CudaCheck(hipStreamCreate(&((*ctx)->send_stream))); });
    (*ctx)->num_block = MAX_NUM_BLOCK;
    NumaAwareCudaMallocHost((*ctx)->dst_dev_id, reinterpret_cast<void**>(&((*ctx)->recv_cnt_ptr)),
                            sizeof(int32_t) * (*ctx)->num_block);
    NumaAwareCudaMallocHost((*ctx)->dst_dev_id, reinterpret_cast<void**>(&((*ctx)->send_cnt_ptr)),
                            sizeof(int32_t) * (*ctx)->num_block);
    *((*ctx)->recv_cnt_ptr) = 0;
    *((*ctx)->send_cnt_ptr) = 0;
    (*ctx)->buf_cap = DEFAULT_CHUNK_BUF_CAP;
    NumaAwareCudaMallocHost((*ctx)->dst_dev_id, reinterpret_cast<void**>(&((*ctx)->buf_ptr)),
                            CHUNK_SIZE * (*ctx)->buf_cap * (*ctx)->num_block);
    CHECK_EQ(reinterpret_cast<std::uintptr_t>((*ctx)->buf_ptr) % PACK_ALIGN, 0);
  }
}

void CudaCopyPeerKernelUtil::CtxDestroy(CudaCopyPeerCtx* ctx) {
  if (!ctx->p2p_enabled) {
    WithCudaDevice(ctx->src_dev_id, [ctx]() {
      CudaCheck(hipStreamSynchronize(ctx->send_stream));
      CudaCheck(hipStreamDestroy(ctx->send_stream));
    });
    CudaCheck(hipHostFree(ctx->recv_cnt_ptr));
    CudaCheck(hipHostFree(ctx->send_cnt_ptr));
    CudaCheck(hipHostFree(ctx->buf_ptr));
  }
  delete ctx;
}

void CudaCopyPeerKernelUtil::CopyAsync(CudaCopyPeerCtx* ctx, void* dst, const void* src,
                                       int32_t size) {
  if (ctx->p2p_enabled) {
    CHECK_EQ(size % PACK_SIZE, 0);
    CHECK_EQ(reinterpret_cast<std::uintptr_t>(dst) % PACK_ALIGN, 0);
    CHECK_EQ(reinterpret_cast<std::uintptr_t>(src) % PACK_ALIGN, 0);
  } else {
    CHECK_EQ(size % PACK_SIZE, 0);
    CHECK_EQ(reinterpret_cast<std::uintptr_t>(dst) % PACK_ALIGN, 0);
    CHECK_EQ(reinterpret_cast<std::uintptr_t>(src) % PACK_ALIGN, 0);
    WithCudaDevice(ctx->src_dev_id, [&]() {
      Copy<<<ctx->num_block, NUM_THREAD, 0, ctx->send_stream>>>(
          dst, src, size, ctx->buf_ptr, ctx->buf_cap, ctx->send_cnt_ptr, ctx->recv_cnt_ptr, true);
    });
    WithCudaDevice(ctx->dst_dev_id, [&]() {
      Copy<<<ctx->num_block, NUM_THREAD, 0, ctx->recv_stream>>>(
          dst, src, size, ctx->buf_ptr, ctx->buf_cap, ctx->send_cnt_ptr, ctx->recv_cnt_ptr, false);
    });
    hipLaunchParams params[2];
    params[0].func = params[1].func = (void*)Null;
    params[0].gridDim = params[1].gridDim = {1, 1, 1};
    params[0].blockDim = params[1].blockDim = {1, 1, 1};
    params[0].sharedMem = params[1].sharedMem = 0;
    void* args[] = {};
    params[0].args = params[1].args = args;
    params[0].stream = ctx->send_stream;
    params[1].stream = ctx->recv_stream;
    CudaCheck(hipLaunchCooperativeKernelMultiDevice(params, 2));
  }
}

}  // namespace oneflow
