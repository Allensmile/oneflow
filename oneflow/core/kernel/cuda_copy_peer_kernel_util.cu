#include "hip/hip_runtime.h"
#include "oneflow/core/kernel/cuda_copy_peer_kernel_util.h"
#include "oneflow/core/kernel/kernel_util.cuh"

constexpr int32_t PACK_SIZE = sizeof(ulong2);
constexpr int32_t PACK_ALIGN = alignof(ulong2);
constexpr int32_t NUM_THREAD = 1024;
constexpr int32_t NUM_STEP_PER_CHUNK = 16;
constexpr int32_t STEP_SIZE = NUM_THREAD * PACK_SIZE;
constexpr int32_t CHUNK_SIZE = STEP_SIZE * NUM_STEP_PER_CHUNK;
constexpr int32_t DEFAULT_CHUNK_BUF_CAP = 2;
constexpr int32_t WARP_SIZE = 32;
constexpr int32_t MAX_NUM_BLOCK = 2;

namespace oneflow {

namespace {

__forceinline__ __device__ int32_t DivUp(int32_t n, int32_t val) { return (n + val - 1) / val; }

__forceinline__ __device__ void Fetch(ulong2& v, const ulong2* p) {
  // clang-format off
  asm volatile("ld.volatile.global.v2.u64 {%0,%1}, [%2];" : "=l"(v.x), "=l"(v.y) : "l"(p) : "memory");
  // clang-format on
}

__forceinline__ __device__ void Store(ulong2* p, ulong2& v) {
  // clang-format off
  asm volatile("st.volatile.global.v2.u64 [%0], {%1,%2};" :: "l"(p), "l"(v.x), "l"(v.y) : "memory");
  // clang-format on
}

__forceinline__ __device__ void FetchStore(ulong2* dst, const ulong2* src) {
  ulong2 v;
  Fetch(v, src);
  Store(dst, v);
}

__forceinline__ __device__ void FetchStore(void* dst, const void* src) {
  FetchStore(reinterpret_cast<ulong2*>(dst), reinterpret_cast<const ulong2*>(src));
}

__forceinline__ __device__ void CopyChunk(void* dst, const void* src, const int32_t thread_id) {
  dst = (unsigned char*)(dst) + thread_id * PACK_SIZE;
  src = (unsigned char*)(src) + thread_id * PACK_SIZE;
#pragma unroll
  for (int32_t i = 0; i < NUM_STEP_PER_CHUNK; ++i) {
    FetchStore(dst, src);
    dst = (unsigned char*)(dst) + STEP_SIZE;
    src = (const unsigned char*)(src) + STEP_SIZE;
  }
}

__forceinline__ __device__ void CopyPartialChunk(void* dst, const void* src, const int32_t size,
                                                 const int32_t thread_id) {
  int32_t offset = thread_id * PACK_SIZE;
  for (int32_t i = 0; i < NUM_STEP_PER_CHUNK; ++i) {
    if (offset < size) {
      FetchStore((unsigned char*)(dst) + offset, (const unsigned char*)(src) + offset);
    }
    offset += STEP_SIZE;
  }
}

__forceinline__ __device__ void Send(const void* src, const int32_t size, const int32_t thread_id,
                                     void* buf_ptr, const int32_t buf_cap,
                                     volatile int32_t* send_cnt_ptr,
                                     volatile int32_t* recv_cnt_ptr) {
  const int32_t num_chunk = DivUp(size, CHUNK_SIZE);
  int32_t remaining = size;
  if (thread_id == 0) {
    while (*recv_cnt_ptr != 0) {}
    while (*send_cnt_ptr != 0) {}
  }
  __syncthreads();
  for (int32_t chunk = 0; chunk < num_chunk; ++chunk) {
    if (thread_id == WARP_SIZE) {
      while (chunk - *recv_cnt_ptr >= buf_cap) {}
    }
    __syncthreads();
    void* cur_buf_ptr = (unsigned char*)buf_ptr + (chunk % buf_cap) * CHUNK_SIZE;
    if (remaining >= CHUNK_SIZE) {
      CopyChunk(cur_buf_ptr, src, thread_id);
    } else {
      CopyPartialChunk(cur_buf_ptr, src, remaining, thread_id);
    }
    remaining -= CHUNK_SIZE;
    src = (const unsigned char*)(src) + CHUNK_SIZE;
    __threadfence_system();
    __syncthreads();
    if (thread_id == 0) { *send_cnt_ptr = chunk + 1; }
  }
}

__forceinline__ __device__ void Recv(void* dst, const int32_t size, const int32_t thread_id,
                                     const void* buf_ptr, const int32_t buf_cap,
                                     volatile int32_t* send_cnt_ptr,
                                     volatile int32_t* recv_cnt_ptr) {
  const int32_t num_chunk = DivUp(size, CHUNK_SIZE);
  int32_t remaining = size;
  for (int32_t chunk = 0; chunk < num_chunk; ++chunk) {
    if (thread_id == WARP_SIZE) {
      while (*send_cnt_ptr <= chunk) {}
    }
    __syncthreads();
    void* cur_buf_ptr = (unsigned char*)buf_ptr + (chunk % buf_cap) * CHUNK_SIZE;
    if (remaining >= CHUNK_SIZE) {
      CopyChunk(dst, cur_buf_ptr, thread_id);
    } else {
      CopyPartialChunk(dst, cur_buf_ptr, remaining, thread_id);
    }
    remaining -= CHUNK_SIZE;
    dst = (unsigned char*)(dst) + CHUNK_SIZE;
    __syncthreads();
    if (thread_id == 0) { *recv_cnt_ptr = chunk + 1; }
  }
  if (thread_id == 0) {
    *recv_cnt_ptr = 0;
    *send_cnt_ptr = 0;
  }
}

__launch_bounds__(NUM_THREAD) __global__
    void Copy(void* dst, const void* src, const int32_t size, void* buf_ptr, const int32_t buf_cap,
              int32_t* send_cnt_ptr, int32_t* recv_cnt_ptr, bool send_or_recv) {
  const int32_t block_id = blockIdx.x;
  const int32_t num_block = gridDim.x;
  const int32_t thread_id = threadIdx.x;
  const int32_t block_size = DivUp(size / PACK_SIZE, num_block) * PACK_SIZE;
  void* this_block_dst = reinterpret_cast<unsigned char*>(dst) + block_size * block_id;
  const void* this_block_src = reinterpret_cast<const unsigned char*>(src) + block_size * block_id;
  const int32_t this_block_size =
      (block_id + 1) * block_size <= size ? block_size : max(0, size - block_id * block_size);
  void* this_buf_ptr = reinterpret_cast<unsigned char*>(buf_ptr) + CHUNK_SIZE * buf_cap * block_id;
  int32_t* this_send_cnt_ptr = send_cnt_ptr + block_id;
  int32_t* this_recv_cnt_ptr = recv_cnt_ptr + block_id;
  if (send_or_recv) {
    Send(this_block_src, this_block_size, thread_id, this_buf_ptr, buf_cap, this_send_cnt_ptr,
         this_recv_cnt_ptr);
  } else {
    Recv(this_block_dst, this_block_size, thread_id, this_buf_ptr, buf_cap, this_send_cnt_ptr,
         this_recv_cnt_ptr);
  }
}

}  // namespace

struct CudaCopyPeerCtx {
  int32_t dst_dev_id;
  int32_t src_dev_id;
  hipStream_t recv_stream;
  hipStream_t send_stream;
  hipEvent_t sync_event;
  int32_t num_block;
  int32_t* recv_cnt_ptr;
  int32_t* send_cnt_ptr;
  void* buf_ptr;
  int32_t buf_cap;
  bool p2p_enabled;
};

void CudaCopyPeerKernelUtil::CtxCreate(CudaCopyPeerCtx** ctx, int32_t dst_dev_id,
                                       int32_t src_dev_id, hipStream_t recv_stream) {
  *ctx = new CudaCopyPeerCtx();
  (*ctx)->dst_dev_id = dst_dev_id;
  (*ctx)->src_dev_id = src_dev_id;
  (*ctx)->recv_stream = recv_stream;

  WithCudaDevice(dst_dev_id, [ctx]() {
    int32_t can_access;
    CudaCheck(hipDeviceCanAccessPeer(&can_access, (*ctx)->dst_dev_id, (*ctx)->src_dev_id));
    if (can_access) {
      hipError_t error = hipDeviceEnablePeerAccess((*ctx)->src_dev_id, 0);
      if (error != hipErrorPeerAccessAlreadyEnabled) { CudaCheck(error); }
      (*ctx)->p2p_enabled = true;
    } else {
      (*ctx)->p2p_enabled = false;
    }
  });
  if (!(*ctx)->p2p_enabled) {
    WithCudaDevice(src_dev_id, [ctx]() { CudaCheck(hipStreamCreate(&((*ctx)->send_stream))); });
    WithCudaDevice(dst_dev_id, [ctx]() {
      CudaCheck(hipEventCreateWithFlags(&((*ctx)->sync_event),
                                         hipEventBlockingSync | hipEventDisableTiming));
    });
    (*ctx)->num_block = MAX_NUM_BLOCK;
    NumaAwareCudaMallocHost((*ctx)->dst_dev_id, reinterpret_cast<void**>(&((*ctx)->recv_cnt_ptr)),
                            sizeof(int32_t) * (*ctx)->num_block);
    NumaAwareCudaMallocHost((*ctx)->dst_dev_id, reinterpret_cast<void**>(&((*ctx)->send_cnt_ptr)),
                            sizeof(int32_t) * (*ctx)->num_block);
    memset((*ctx)->recv_cnt_ptr, 0, sizeof(int32_t) * (*ctx)->num_block);
    memset((*ctx)->send_cnt_ptr, 0, sizeof(int32_t) * (*ctx)->num_block);
    (*ctx)->buf_cap = DEFAULT_CHUNK_BUF_CAP;
    NumaAwareCudaMallocHost((*ctx)->dst_dev_id, reinterpret_cast<void**>(&((*ctx)->buf_ptr)),
                            CHUNK_SIZE * (*ctx)->buf_cap * (*ctx)->num_block);
    CHECK_EQ(reinterpret_cast<std::uintptr_t>((*ctx)->buf_ptr) % PACK_ALIGN, 0);
  }
}

void CudaCopyPeerKernelUtil::CtxDestroy(CudaCopyPeerCtx* ctx) {
  if (!ctx->p2p_enabled) {
    WithCudaDevice(ctx->src_dev_id, [ctx]() {
      CudaCheck(hipStreamSynchronize(ctx->send_stream));
      CudaCheck(hipStreamDestroy(ctx->send_stream));
    });
    WithCudaDevice(ctx->dst_dev_id, [ctx]() { CudaCheck(hipEventDestroy(ctx->sync_event)); });
    CudaCheck(hipHostFree(ctx->recv_cnt_ptr));
    CudaCheck(hipHostFree(ctx->send_cnt_ptr));
    CudaCheck(hipHostFree(ctx->buf_ptr));
  }
  delete ctx;
}

void CudaCopyPeerKernelUtil::CopyAsync(CudaCopyPeerCtx* ctx, void* dst, const void* src,
                                       int32_t size) {
  if (ctx->p2p_enabled) {
    CudaCheck(hipMemcpyAsync(dst, src, size, hipMemcpyDefault, ctx->recv_stream));
  } else {
    CHECK_EQ(size % PACK_SIZE, 0);
    CHECK_EQ(reinterpret_cast<std::uintptr_t>(dst) % PACK_ALIGN, 0);
    CHECK_EQ(reinterpret_cast<std::uintptr_t>(src) % PACK_ALIGN, 0);
    WithCudaDevice(ctx->dst_dev_id, [&]() {
      CudaCheck(hipEventRecord(ctx->sync_event, ctx->recv_stream));
      Copy<<<ctx->num_block, NUM_THREAD, 0, ctx->recv_stream>>>(
          dst, src, size, ctx->buf_ptr, ctx->buf_cap, ctx->send_cnt_ptr, ctx->recv_cnt_ptr, false);
    });
    WithCudaDevice(ctx->src_dev_id, [&]() {
      CudaCheck(hipStreamWaitEvent(ctx->send_stream, ctx->sync_event, 0));
      Copy<<<ctx->num_block, NUM_THREAD, 0, ctx->send_stream>>>(
          dst, src, size, ctx->buf_ptr, ctx->buf_cap, ctx->send_cnt_ptr, ctx->recv_cnt_ptr, true);
    });
  }
}

}  // namespace oneflow
